#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 128
#define BLOCK_STEP 32
#define BLOCK_NUM 32


#define MOD(a,b) ((a) - (a) / (b) * (b))

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__ void bodyForce(Body *p,int *flag,float dt, int n) {
        //每个线程算一个点的一部分
                int i=MOD(threadIdx.x+blockIdx.x*blockDim.x,n);
        int nn=n/(BLOCK_STEP*BLOCK_SIZE);
        float Fx = 0.0f;
                float Fy = 0.0f;
                float Fz = 0.0f;
                __shared__ float3 p_shared[BLOCK_SIZE];
                float xi=p[i].x;
                float yi=p[i].y;
                float zi=p[i].z;//储存对应位置的数据
                float dx,dy,dz,distSqr,invDist,invDist3;

                int loc;
                #pragma unroll 8
                for(int j=(blockIdx.x+blockIdx.x/BLOCK_NUM);nn-->0;j+=BLOCK_STEP){
                        //从当前块开始向前推进
                        j=MOD(j,BLOCK_NUM);
                        //获得当前块中的当前相对位置的数据，并写到shared中
                        loc=j*BLOCK_SIZE+threadIdx.x;
                        p_shared[threadIdx.x]=make_float3(p[loc].x,p[loc].y,p[loc].z);
                        //同步锁
                        __syncthreads();
                        #pragma unroll 8
                        for(int k=0;k<BLOCK_SIZE;k++){
                                        dx = p_shared[k].x - xi;
                                        dy = p_shared[k].y - yi;
                                        dz = p_shared[k].z - zi;
                                        distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                                        invDist = rsqrtf(distSqr);
                                        invDist3 = invDist * invDist * invDist;
                                        Fx += dx * invDist3;
                                        Fy += dy * invDist3;
                                        Fz += dz * invDist3;
                        }
                        //算完之后才能写入新数据
                        __syncthreads();
                }
                //原子加保证数据正确性
                atomicAdd(&p[i].vx, dt * Fx);
                atomicAdd(&p[i].vy, dt * Fy);
                atomicAdd(&p[i].vz, dt * Fz);

                atomicSub(&flag[i], 1);
                if(!atomicMax(&flag[i], 0)){
                        // integrate position
                        atomicAdd(&p[i].x,p[i].vx*dt);
                        atomicAdd(&p[i].y,p[i].vy*dt);
                        atomicAdd(&p[i].z,p[i].vz*dt);
                        atomicExch(&flag[i],BLOCK_STEP);
                }

}

int main(const int argc, const char** argv) {

  /*
   * Do not change the value for `nBodies` here. If you would like to modify it,
   * pass values into the command line.
   */

  int nBodies = 2<<11;
  int salt = 1;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

  if (argc > 2) salt = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;
  hipHostMalloc((void **)&buf,bytes);

  /*
   * As a constraint of this exercise, `randomizeBodies` must remain a host function.
   */
  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data
  float *d_buf;
  hipMalloc((void **)&d_buf,bytes);
  Body *d_p=(Body *)d_buf;
  hipMemcpy(d_buf,buf,bytes,hipMemcpyHostToDevice);

  int *flag;
  hipHostMalloc(&flag,nBodies*sizeof(int));
  for(int i=0;i<nBodies;i++)
        flag[i]=BLOCK_STEP;
  int *d_flag;
  hipMalloc(&d_flag,nBodies*sizeof(int));
  hipMemcpy(d_flag,flag,nBodies*sizeof(int),hipMemcpyHostToDevice);


  double totalTime = 0.0;

  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

  /*******************************************************************/
  // Do not modify these 2 lines of code.
  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();
  /*******************************************************************/

  /*
   * You will likely wish to refactor the work being done in `bodyForce`,
   * as well as the work to integrate the positions.
   */

    bodyForce<<<BLOCK_NUM*BLOCK_STEP,BLOCK_SIZE>>>(d_p,d_flag,dt, nBodies); // compute interbody forces

  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */

   if(iter==nIters-1)
       hipMemcpy(buf,d_buf,bytes,hipMemcpyDeviceToHost);

  /*******************************************************************/
  // Do not modify the code in this section.
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }



  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
  checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
  checkAccuracy(buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/

  /*
   * Feel free to modify code below.
   */
  hipFree(buf);
  hipFree(flag);
}
